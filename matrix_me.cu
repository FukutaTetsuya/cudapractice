#include "hip/hip_runtime.h"
/*
 *C_ij =  C[i * n + j]とする
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define NT 32//NT*NT threads per block
#define NB 8//blocks to use
#define N 600//N*N square matrix
__device__ __constant__ int n;//n = N

//GPU functions-----------------------------------------------------------------
__global__ void matrix_product_global(double *A_device, double *B_device, double *C_device) {
	//1 thread 1 C element
	int i_global, j_global;
	int i, j, k;
	double temp;

	i_global = blockDim.x * blockIdx.x + threadIdx.x;
	j_global = threadIdx.y;

	for(i = i_global; i < n; i += NB * NT) {
		for(j = j_global; j < n; j += NT) {
			temp = 0.0;
			for(k = 0; k < n; k += 1) {
				temp += A_device[i * n + k] * B_device[k * n + j];
			}
			C_device[i * n + j] = temp;
		}
	}
}

__global__ void matrix_product_shared(double *A_device, double *B_device, double *C_device) {
	//devide A,B,C into small blocks
	//arbitrary dimensional matrix
	int i_global, j_global;
	int i_local, j_local;
	int i_offset, j_offset;
	int i, j, k;
	__shared__ double a[NT * NT];
	__shared__ double b[NT * NT];
	i_local = threadIdx.x;
	j_local = threadIdx.y;
	for(i_offset = 0; i_goffset < n; i_goffset += NT * NB) {
		for(j_offset = 0; j_offset < n; j_offset += NT) {
			//load global to shared
			for(k = 0; k < NT; k += 1) {
			}
		}
	}
}

//Host functions----------------------------------------------------------------
void init_matrix(double *matrix, int n_host) {
	int i;
	int j;
	for(i = 0; i < n_host; i += 1) {
		for(j = 0; j < n_host; j += 1) {
			//matrix[i * n_host + j] = (double)rand() / (double)RAND_MAX;
			matrix[i * n_host + j] = 1.0;
		}
	}
}
void matrix_product_host(double *A, double *B, double *C, int n_host) {
	int i;
	int j;
	int k;
	double temporal;
	for(i = 0; i < n_host; i += 1) {
		for(j = 0; j < n_host; j += 1) {
			//ij loop for each component of matrix C
			temporal = 0.0;
			for(k = 0; k < n_host; k += 1) {
				temporal += A[i * n_host + k] * B[k * n_host + j];
			}
			C[i * n_host + j] = temporal;
		}
	}
}

double calc_deviation(double *M1, double *M2, int n_host) {
	int i;
	double devi = 0.0;
	for(i = 0; i < n_host; i += 1) {
		devi += (M1[i] - M2[i]) * (M1[i] - M2[i]);
	}
	return devi;
}

int main(void) {
	int n_host;
	int n_square;
	dim3 dim_threads;
	double *A_device;
	double *B_device;
	double *C_device;
	double *A_host;
	double *B_host;
	double *C_host;
	double *result_host;
	double *result_global;
	double *result_shared;
	clock_t start, end;
//initialize--------------------------------------------------------------------
	//constants-------------------------------------------------------------
	n_host = N;
	n_square = n_host * n_host;
	dim_threads.x = NT;
	dim_threads.y = NT;
	dim_threads.z = 1;
	hipMemcpyToSymbol(HIP_SYMBOL(n), &n_host, sizeof(int), 0, hipMemcpyHostToDevice);

	//allocate--------------------------------------------------------------
	hipHostAlloc((void **)&A_host, n_square * sizeof(double), hipHostMallocMapped);
	hipHostAlloc((void **)&B_host, n_square * sizeof(double), hipHostMallocMapped);
	hipHostAlloc((void **)&C_host, n_square * sizeof(double), hipHostMallocMapped);
	result_host = (double *)calloc(n_square, sizeof(double));
	result_global = (double *)calloc(n_square, sizeof(double));

	hipMalloc((void **)&A_device, n_square * sizeof(double));
	hipMalloc((void **)&B_device, n_square * sizeof(double));
	hipMalloc((void **)&C_device, n_square * sizeof(double));

	//init matrix------------------------------------------------------------
	init_matrix(A_host, n_host);
	init_matrix(B_host, n_host);
	hipMemcpy(A_device, A_host, n_square * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(B_device, B_host, n_square * sizeof(double), hipMemcpyHostToDevice);

//calculate---------------------------------------------------------------------
	//host------------------------------------------------------------------
	start = clock();
	matrix_product_host(A_host, B_host, C_host, n_host);
	end = clock();
	memcpy(result_host, C_host, n_square * sizeof(double));
	printf("%d [ms]\n", (int)(1000*(end - start)/CLOCKS_PER_SEC));

	//global----------------------------------------------------------------
	start = clock();
	matrix_product_global<<<NB, dim_threads>>>(A_device, B_device, C_device);
	hipDeviceSynchronize();
	end = clock();
	hipMemcpy(result_global, C_device, n_square * sizeof(double), hipMemcpyDeviceToHost);
	printf("%d [ms]\n", (int)(1000*(end - start)/CLOCKS_PER_SEC));

//check the answers-------------------------------------------------------------
	printf("check\n");
	printf("global:%f\n", calc_deviation(result_host, result_global, n_host));

//finalize----------------------------------------------------------------------
	hipHostFree(A_host);
	hipHostFree(B_host);
	hipHostFree(C_host);
	free(result_host);

	hipFree(A_device);
	hipFree(B_device);
	hipFree(C_device);
	return 0;
}	

