
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

/*  square root of number of threads in a block (the number of threads in a block is NT^2)  */
#define NT 32
/*  length of the target domain  */
#define L 10.0
/*  number of division for the discretization of the target domain  */
#define N 256
/*  dimensionless time step size (theta = D * dt / dx^2)  */
#define THETA 0.1
/*  number of iterations  */
#define M 2000


/*  constants on a GPU  */
__device__ __constant__ int n;
__device__ __constant__ float theta;

//GPU functions-----------------------------------------------------------------

//Host functions----------------------------------------------------------------
void init_field(double *field_host, int n_host, int l_host) {
	int i;
	int j;
	double x;
	double y;
	double dx = l_host / (double)n_host;
	double dy = l_host / (double)n_host;
	double midst = l_host * 0.5;
	for(i = 0; i < n_host; i += 1) {
		x = (double)i * dx;
		for(j = 0; j < n_host; j += 1) {
			y = (double)j * dy;
			if((x > midst && y > midst) || (x < midst && y < midst)) {
				field_host[n_host * j + i] = 1.0;
			} else {
				field_host[n_host * j + i] = 0.0;
			}
		}
	}
}

int main(void) {
//variavles---------------------------------------------------------------------
	int n_host;
	int n_square;
	int iteration;
	int n_blocks;
	double l_host;
	double theta_host;
	dim3 dim_threads;
	double *field_host[2];
	double *field_device[2];
	double *result_global_host;
	double *result_shared_host;
	FILE *file_write;

//initialize--------------------------------------------------------------------
	//set variables---------------------------------------------------------
	n_host = N;
	n_square = N * N;
	l_host = L;
	dim_threads.x = NT;
	dim_threads.y = NT;
	dim_threads.z = 1;
	n_blocks = (int)(ceil((double)n_host / NT));
	iteration = M;

	//allocate memories-----------------------------------------------------
	hipMemcpyToSymbol(HIP_SYMBOL(n), &n_host, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(theta), &theta_host, sizeof(double), 0, hipMemcpyHostToDevice);
	hipHostAlloc((void **)&field_host[0], n_square * sizeof(double), hipHostMallocMapped);
	hipHostAlloc((void **)&field_host[1], n_square * sizeof(double), hipHostMallocMapped);
	hipHostAlloc((void **)&result_global_host,  n_square * sizeof(double), hipHostMallocMapped);
	hipHostAlloc((void **)&result_shared_host,  n_square * sizeof(double), hipHostMallocMapped);
	hipMalloc((void **)&field_device[0], n_square * sizeof(double));
	hipMalloc((void **)&field_device[1], n_square * sizeof(double));

	//initialize field------------------------------------------------------
	init_field(field_host[0], n_host, l_host);
	hipMemcpy(field_device[0], field_host[0], n_square * sizeof(double), hipMemcpyHostToDevice);
//finalize----------------------------------------------------------------------
	hipHostFree(field_host[0]);
	hipHostFree(field_host[1]);
	hipHostFree(result_global_host);
	hipHostFree(result_shared_host);
	hipFree(field_device[0]);
	hipFree(field_device[1]);

	return 0;
}
