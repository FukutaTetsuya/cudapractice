
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

/*  square root of number of threads in a block (the number of threads in a block is NT^2)  */
#define NT 32
/*  length of the target domain  */    
#define L 10.0
/*  number of division for the discretization of the target domain  */
#define N 256
/*  dimensionless time step size (theta = D * dt / dx^2)  */
#define THETA 0.1
/*  number of iterations  */
#define M 2000


/*  constants on a GPU  */
__device__ __constant__ int n;
__device__ __constant__ float theta;



